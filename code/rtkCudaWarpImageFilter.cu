#include "hip/hip_runtime.h"
/*=========================================================================
 *
 *  Copyright RTK Consortium
 *
 *  Licensed under the Apache License, Version 2.0 (the "License");
 *  you may not use this file except in compliance with the License.
 *  You may obtain a copy of the License at
 *
 *         http://www.apache.org/licenses/LICENSE-2.0.txt
 *
 *  Unless required by applicable law or agreed to in writing, software
 *  distributed under the License is distributed on an "AS IS" BASIS,
 *  WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 *  See the License for the specific language governing permissions and
 *  limitations under the License.
 *
 *=========================================================================*/

/* -----------------------------------------------------------------------
   See COPYRIGHT.TXT and LICENSE.TXT for copyright and license information
   ----------------------------------------------------------------------- */
/*****************
*  rtk #includes *
*****************/
#include "rtkCudaUtilities.hcu"
#include "rtkConfiguration.h"
#include "rtkCudaWarpImageFilter.hcu"

/*****************
*  C   #includes *
*****************/
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <math.h>

/*****************
* CUDA #includes *
*****************/
#include <hip/hip_runtime.h>

// T E X T U R E S ////////////////////////////////////////////////////////
texture<float, 1, hipReadModeElementType> tex_IndexOutputToPPOutputMatrix;
texture<float, 1, hipReadModeElementType> tex_IndexOutputToIndexDVFMatrix;
texture<float, 1, hipReadModeElementType> tex_PPInputToIndexInputMatrix;

texture<float, 3, hipReadModeElementType> tex_xdvf;
texture<float, 3, hipReadModeElementType> tex_ydvf;
texture<float, 3, hipReadModeElementType> tex_zdvf;
texture<float, 3, hipReadModeElementType> tex_input_vol;
///////////////////////////////////////////////////////////////////////////

//_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_
// K E R N E L S -_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_
//_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_( S T A R T )_
//_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_

__global__
void kernel(float *dev_vol_out, int3 vol_dim, unsigned int Blocks_Y)
{
  // CUDA 2.0 does not allow for a 3D grid, which severely
  // limits the manipulation of large 3D arrays of data.  The
  // following code is a hack to bypass this implementation
  // limitation.
  unsigned int blockIdx_z = blockIdx.y / Blocks_Y;
  unsigned int blockIdx_y = blockIdx.y - __umul24(blockIdx_z, Blocks_Y);
  unsigned int i = __umul24(blockIdx.x, blockDim.x) + threadIdx.x;
  unsigned int j = __umul24(blockIdx_y, blockDim.y) + threadIdx.y;
  unsigned int k = __umul24(blockIdx_z, blockDim.z) + threadIdx.z;

  if (i >= vol_dim.x || j >= vol_dim.y || k >= vol_dim.z)
    {
    return;
    }

  // Index row major into the volume
  long int vol_idx = i + (j + k*vol_dim.y)*(vol_dim.x);

  // Matrix multiply to get the index in the DVF texture of the current point in the output volume
  float3 IndexInDVF;
  IndexInDVF.x = tex1Dfetch(tex_IndexOutputToIndexDVFMatrix, 0)*i + tex1Dfetch(tex_IndexOutputToIndexDVFMatrix, 1)*j +
         tex1Dfetch(tex_IndexOutputToIndexDVFMatrix, 2)*k + tex1Dfetch(tex_IndexOutputToIndexDVFMatrix, 3);
  IndexInDVF.y = tex1Dfetch(tex_IndexOutputToIndexDVFMatrix, 4)*i + tex1Dfetch(tex_IndexOutputToIndexDVFMatrix, 5)*j +
         tex1Dfetch(tex_IndexOutputToIndexDVFMatrix, 6)*k + tex1Dfetch(tex_IndexOutputToIndexDVFMatrix, 7);
  IndexInDVF.z = tex1Dfetch(tex_IndexOutputToIndexDVFMatrix, 8)*i + tex1Dfetch(tex_IndexOutputToIndexDVFMatrix, 9)*j +
         tex1Dfetch(tex_IndexOutputToIndexDVFMatrix, 10)*k + tex1Dfetch(tex_IndexOutputToIndexDVFMatrix, 11);

  // Get each component of the displacement vector by
  // interpolation in the dvf
  float3 Displacement;
  Displacement.x = tex3D(tex_xdvf, IndexInDVF.x + 0.5f, IndexInDVF.y + 0.5f, IndexInDVF.z + 0.5f);
  Displacement.y = tex3D(tex_ydvf, IndexInDVF.x + 0.5f, IndexInDVF.y + 0.5f, IndexInDVF.z + 0.5f);
  Displacement.z = tex3D(tex_zdvf, IndexInDVF.x + 0.5f, IndexInDVF.y + 0.5f, IndexInDVF.z + 0.5f);

  // Matrix multiply to get the physical coordinates of the current point in the output volume
  float3 PPinOutput;
  PPinOutput.x = tex1Dfetch(tex_IndexOutputToPPOutputMatrix, 0)*i + tex1Dfetch(tex_IndexOutputToPPOutputMatrix, 1)*j +
               tex1Dfetch(tex_IndexOutputToPPOutputMatrix, 2)*k + tex1Dfetch(tex_IndexOutputToPPOutputMatrix, 3);
  PPinOutput.y = tex1Dfetch(tex_IndexOutputToPPOutputMatrix, 4)*i + tex1Dfetch(tex_IndexOutputToPPOutputMatrix, 5)*j +
               tex1Dfetch(tex_IndexOutputToPPOutputMatrix, 6)*k + tex1Dfetch(tex_IndexOutputToPPOutputMatrix, 7);
  PPinOutput.z = tex1Dfetch(tex_IndexOutputToPPOutputMatrix, 8)*i + tex1Dfetch(tex_IndexOutputToPPOutputMatrix, 9)*j +
               tex1Dfetch(tex_IndexOutputToPPOutputMatrix, 10)*k + tex1Dfetch(tex_IndexOutputToPPOutputMatrix, 11);

  // Get the index corresponding to the current physical point in output displaced by the displacement vector
  float3 PPDisplaced;
  PPDisplaced.x = PPinOutput.x + Displacement.x;
  PPDisplaced.y = PPinOutput.y + Displacement.y;
  PPDisplaced.z = PPinOutput.z + Displacement.z;

  float3 IndexInInput;
  IndexInInput.x =  tex1Dfetch(tex_PPInputToIndexInputMatrix, 0) * PPDisplaced.x
                  + tex1Dfetch(tex_PPInputToIndexInputMatrix, 1) * PPDisplaced.y
                  + tex1Dfetch(tex_PPInputToIndexInputMatrix, 2) * PPDisplaced.z
                  + tex1Dfetch(tex_PPInputToIndexInputMatrix, 3);
  IndexInInput.y =  tex1Dfetch(tex_PPInputToIndexInputMatrix, 4) * PPDisplaced.x
                  + tex1Dfetch(tex_PPInputToIndexInputMatrix, 5) * PPDisplaced.y
                  + tex1Dfetch(tex_PPInputToIndexInputMatrix, 6) * PPDisplaced.z
                  + tex1Dfetch(tex_PPInputToIndexInputMatrix, 7);
  IndexInInput.z =  tex1Dfetch(tex_PPInputToIndexInputMatrix, 8) * PPDisplaced.x
                  + tex1Dfetch(tex_PPInputToIndexInputMatrix, 9) * PPDisplaced.y
                  + tex1Dfetch(tex_PPInputToIndexInputMatrix, 10)* PPDisplaced.z
                  + tex1Dfetch(tex_PPInputToIndexInputMatrix, 11);

  // Interpolate in the input and copy into the output
  dev_vol_out[vol_idx] = tex3D(tex_input_vol, IndexInInput.x + 0.5f, IndexInInput.y + 0.5f, IndexInInput.z + 0.5f);
}

__global__
void kernel_3Dgrid(float * dev_vol_out, int3 vol_dim)
{
  unsigned int i = __umul24(blockIdx.x, blockDim.x) + threadIdx.x;
  unsigned int j = __umul24(blockIdx.y, blockDim.y) + threadIdx.y;
  unsigned int k = __umul24(blockIdx.z, blockDim.z) + threadIdx.z;

  if (i >= vol_dim.x || j >= vol_dim.y || k >= vol_dim.z)
    {
    return;
    }

  // Index row major into the volume
  long int vol_idx = i + (j + k*vol_dim.y)*(vol_dim.x);

  // Matrix multiply to get the index in the DVF texture of the current point in the output volume
  float3 IndexInDVF;
  IndexInDVF.x = tex1Dfetch(tex_IndexOutputToIndexDVFMatrix, 0)*i + tex1Dfetch(tex_IndexOutputToIndexDVFMatrix, 1)*j +
         tex1Dfetch(tex_IndexOutputToIndexDVFMatrix, 2)*k + tex1Dfetch(tex_IndexOutputToIndexDVFMatrix, 3);
  IndexInDVF.y = tex1Dfetch(tex_IndexOutputToIndexDVFMatrix, 4)*i + tex1Dfetch(tex_IndexOutputToIndexDVFMatrix, 5)*j +
         tex1Dfetch(tex_IndexOutputToIndexDVFMatrix, 6)*k + tex1Dfetch(tex_IndexOutputToIndexDVFMatrix, 7);
  IndexInDVF.z = tex1Dfetch(tex_IndexOutputToIndexDVFMatrix, 8)*i + tex1Dfetch(tex_IndexOutputToIndexDVFMatrix, 9)*j +
         tex1Dfetch(tex_IndexOutputToIndexDVFMatrix, 10)*k + tex1Dfetch(tex_IndexOutputToIndexDVFMatrix, 11);

  // Get each component of the displacement vector by
  // interpolation in the dvf
  float3 Displacement;
  Displacement.x = tex3D(tex_xdvf, IndexInDVF.x + 0.5f, IndexInDVF.y + 0.5f, IndexInDVF.z + 0.5f);
  Displacement.y = tex3D(tex_ydvf, IndexInDVF.x + 0.5f, IndexInDVF.y + 0.5f, IndexInDVF.z + 0.5f);
  Displacement.z = tex3D(tex_zdvf, IndexInDVF.x + 0.5f, IndexInDVF.y + 0.5f, IndexInDVF.z + 0.5f);

  // Matrix multiply to get the physical coordinates of the current point in the output volume
  float3 PPinOutput;
  PPinOutput.x = tex1Dfetch(tex_IndexOutputToPPOutputMatrix, 0)*i + tex1Dfetch(tex_IndexOutputToPPOutputMatrix, 1)*j +
               tex1Dfetch(tex_IndexOutputToPPOutputMatrix, 2)*k + tex1Dfetch(tex_IndexOutputToPPOutputMatrix, 3);
  PPinOutput.y = tex1Dfetch(tex_IndexOutputToPPOutputMatrix, 4)*i + tex1Dfetch(tex_IndexOutputToPPOutputMatrix, 5)*j +
               tex1Dfetch(tex_IndexOutputToPPOutputMatrix, 6)*k + tex1Dfetch(tex_IndexOutputToPPOutputMatrix, 7);
  PPinOutput.z = tex1Dfetch(tex_IndexOutputToPPOutputMatrix, 8)*i + tex1Dfetch(tex_IndexOutputToPPOutputMatrix, 9)*j +
               tex1Dfetch(tex_IndexOutputToPPOutputMatrix, 10)*k + tex1Dfetch(tex_IndexOutputToPPOutputMatrix, 11);

  // Get the index corresponding to the current physical point in output displaced by the displacement vector
  float3 PPDisplaced;
  PPDisplaced.x = PPinOutput.x + Displacement.x;
  PPDisplaced.y = PPinOutput.y + Displacement.y;
  PPDisplaced.z = PPinOutput.z + Displacement.z;

  float3 IndexInInput;
  IndexInInput.x =  tex1Dfetch(tex_PPInputToIndexInputMatrix, 0) * PPDisplaced.x
                  + tex1Dfetch(tex_PPInputToIndexInputMatrix, 1) * PPDisplaced.y
                  + tex1Dfetch(tex_PPInputToIndexInputMatrix, 2) * PPDisplaced.z
                  + tex1Dfetch(tex_PPInputToIndexInputMatrix, 3);
  IndexInInput.y =  tex1Dfetch(tex_PPInputToIndexInputMatrix, 4) * PPDisplaced.x
                  + tex1Dfetch(tex_PPInputToIndexInputMatrix, 5) * PPDisplaced.y
                  + tex1Dfetch(tex_PPInputToIndexInputMatrix, 6) * PPDisplaced.z
                  + tex1Dfetch(tex_PPInputToIndexInputMatrix, 7);
  IndexInInput.z =  tex1Dfetch(tex_PPInputToIndexInputMatrix, 8) * PPDisplaced.x
                  + tex1Dfetch(tex_PPInputToIndexInputMatrix, 9) * PPDisplaced.y
                  + tex1Dfetch(tex_PPInputToIndexInputMatrix, 10)* PPDisplaced.z
                  + tex1Dfetch(tex_PPInputToIndexInputMatrix, 11);

  // Interpolate in the input and copy into the output
  dev_vol_out[vol_idx] = tex3D(tex_input_vol, IndexInInput.x + 0.5f, IndexInInput.y + 0.5f, IndexInInput.z + 0.5f);
}

//_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_
// K E R N E L S -_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_
//_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-( E N D )-_-_
//_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_

///////////////////////////////////////////////////////////////////////////
// FUNCTION: CUDA_warp /////////////////////////////
void
CUDA_warp(int input_vol_dim[3],
    int input_dvf_dim[3],
    int output_vol_dim[3],
    float IndexOutputToPPOutputMatrix[12],
    float IndexOutputToIndexDVFMatrix[12],
    float PPInputToIndexInputMatrix[12],
    float *dev_input_vol,
    float *dev_input_xdvf,
    float *dev_input_ydvf,
    float *dev_input_zdvf,
    float *dev_output_vol,
    bool isLinear)
{

  // Prepare channel description for arrays
  static hipChannelFormatDesc channelDesc = hipCreateChannelDesc<float>();

  ///////////////////////////////////
  // For each component of the dvf, perform a strided copy (pick every third
  // float from dev_input_dvf) into a 3D array, and bind the array to a 3D texture

  // Extent stuff, will be used for each component extraction
  hipExtent dvfExtent = make_hipExtent(input_dvf_dim[0], input_dvf_dim[1], input_dvf_dim[2]);

  // Set texture parameters
  tex_xdvf.addressMode[0] = hipAddressModeBorder;
  tex_xdvf.addressMode[1] = hipAddressModeBorder;
  tex_xdvf.addressMode[2] = hipAddressModeBorder;
  tex_xdvf.filterMode = hipFilterModeLinear;
  tex_xdvf.normalized = false; // don't access with normalized texture coords

  tex_ydvf.addressMode[0] = hipAddressModeBorder;
  tex_ydvf.addressMode[1] = hipAddressModeBorder;
  tex_ydvf.addressMode[2] = hipAddressModeBorder;
  tex_ydvf.filterMode = hipFilterModeLinear;
  tex_ydvf.normalized = false;

  tex_zdvf.addressMode[0] = hipAddressModeBorder;
  tex_zdvf.addressMode[1] = hipAddressModeBorder;
  tex_zdvf.addressMode[2] = hipAddressModeBorder;
  tex_zdvf.filterMode = hipFilterModeLinear;
  tex_zdvf.normalized = false;

  // Allocate the arrays
  hipArray *array_xdvf;
  hipArray *array_ydvf;
  hipArray *array_zdvf;
  hipMalloc3DArray((hipArray**)&array_xdvf, &channelDesc, dvfExtent);
  hipMalloc3DArray((hipArray**)&array_ydvf, &channelDesc, dvfExtent);
  hipMalloc3DArray((hipArray**)&array_zdvf, &channelDesc, dvfExtent);
  CUDA_CHECK_ERROR;

  // Copy image data to arrays. The tricky part is the make_hipPitchedPtr.
  // The best way to understand it is to read
  // http://stackoverflow.com/questions/16119943/how-and-when-should-i-use-pitched-pointer-with-the-cuda-api
  hipMemcpy3DParms xCopyParams = {0};
  xCopyParams.srcPtr   = make_hipPitchedPtr(dev_input_xdvf, input_dvf_dim[0] * sizeof(float), input_dvf_dim[0], input_dvf_dim[1]);
  xCopyParams.dstArray = (hipArray*)array_xdvf;
  xCopyParams.extent   = dvfExtent;
  xCopyParams.kind     = hipMemcpyDeviceToDevice;
  hipMemcpy3D(&xCopyParams);
  CUDA_CHECK_ERROR;

  hipMemcpy3DParms yCopyParams = {0};
  yCopyParams.srcPtr   = make_hipPitchedPtr(dev_input_ydvf, input_dvf_dim[0] * sizeof(float), input_dvf_dim[0], input_dvf_dim[1]);
  yCopyParams.dstArray = (hipArray*)array_ydvf;
  yCopyParams.extent   = dvfExtent;
  yCopyParams.kind     = hipMemcpyDeviceToDevice;
  hipMemcpy3D(&yCopyParams);
  CUDA_CHECK_ERROR;

  hipMemcpy3DParms zCopyParams = {0};
  zCopyParams.srcPtr   = make_hipPitchedPtr(dev_input_zdvf, input_dvf_dim[0] * sizeof(float), input_dvf_dim[0], input_dvf_dim[1]);
  zCopyParams.dstArray = (hipArray*)array_zdvf;
  zCopyParams.extent   = dvfExtent;
  zCopyParams.kind     = hipMemcpyDeviceToDevice;
  hipMemcpy3D(&zCopyParams);
  CUDA_CHECK_ERROR;

  // Bind 3D arrays to 3D textures
  hipBindTextureToArray(tex_xdvf, (hipArray*)array_xdvf, channelDesc);
  hipBindTextureToArray(tex_ydvf, (hipArray*)array_ydvf, channelDesc);
  hipBindTextureToArray(tex_zdvf, (hipArray*)array_zdvf, channelDesc);
  CUDA_CHECK_ERROR;

  ///////////////////////////////////
  // Do the same for the input volume

  // Extent stuff
  hipExtent volExtent = make_hipExtent(input_vol_dim[0], input_vol_dim[1], input_vol_dim[2]);

  // Set texture parameters
  tex_input_vol.addressMode[0] = hipAddressModeBorder;
  tex_input_vol.addressMode[1] = hipAddressModeBorder;
  tex_input_vol.addressMode[2] = hipAddressModeBorder;
  tex_input_vol.normalized = false; // don't access with normalized texture coords
  if (isLinear)
    tex_input_vol.filterMode = hipFilterModeLinear;
  else
    tex_input_vol.filterMode = hipFilterModePoint;

  // Allocate the array
  hipArray *array_input_vol;
  hipMalloc3DArray((hipArray**)&array_input_vol, &channelDesc, volExtent);
  CUDA_CHECK_ERROR;

  // Copy image data to array
  hipMemcpy3DParms inputCopyParams = {0};
  inputCopyParams.srcPtr   = make_hipPitchedPtr(dev_input_vol, input_vol_dim[0]*sizeof(float), input_vol_dim[0], input_vol_dim[1]);
  inputCopyParams.dstArray = (hipArray*)array_input_vol;
  inputCopyParams.extent   = volExtent;
  inputCopyParams.kind     = hipMemcpyDeviceToDevice;
  hipMemcpy3D(&inputCopyParams);
  CUDA_CHECK_ERROR;

  // Bind 3D arrays to 3D textures
  hipBindTextureToArray(tex_input_vol, (hipArray*)array_input_vol, channelDesc);
  CUDA_CHECK_ERROR;

  ///////////////////////////////////////
  // Copy matrices, bind them to textures

  float *dev_IndexOutputToPPOutput;
  hipMalloc( (void**)&dev_IndexOutputToPPOutput, 12*sizeof(float) );
  hipMemcpy (dev_IndexOutputToPPOutput, IndexOutputToPPOutputMatrix, 12*sizeof(float), hipMemcpyHostToDevice);
  hipBindTexture (0, tex_IndexOutputToPPOutputMatrix, dev_IndexOutputToPPOutput, 12*sizeof(float) );

  float *dev_IndexOutputToIndexDVF;
  hipMalloc( (void**)&dev_IndexOutputToIndexDVF, 12*sizeof(float) );
  hipMemcpy (dev_IndexOutputToIndexDVF, IndexOutputToIndexDVFMatrix, 12*sizeof(float), hipMemcpyHostToDevice);
  hipBindTexture (0, tex_IndexOutputToIndexDVFMatrix, dev_IndexOutputToIndexDVF, 12*sizeof(float) );

  float *dev_PPInputToIndexInput;
  hipMalloc( (void**)&dev_PPInputToIndexInput, 12*sizeof(float) );
  hipMemcpy (dev_PPInputToIndexInput, PPInputToIndexInputMatrix, 12*sizeof(float), hipMemcpyHostToDevice);
  hipBindTexture (0, tex_PPInputToIndexInputMatrix, dev_PPInputToIndexInput, 12*sizeof(float) );

  //////////////////////////////////////
  /// Run

  int device;
  hipGetDevice(&device);

  // Thread Block Dimensions
  const int tBlock_x = 16;
  const int tBlock_y = 4;
  const int tBlock_z = 4;

  // Each element in the volume (each voxel) gets 1 thread
  unsigned int  blocksInX = (output_vol_dim[0]-1)/tBlock_x + 1;
  unsigned int  blocksInY = (output_vol_dim[1]-1)/tBlock_y + 1;
  unsigned int  blocksInZ = (output_vol_dim[2]-1)/tBlock_z + 1;

  if(CUDA_VERSION<4000 || GetCudaComputeCapability(device).first<=1)
    {
    dim3 dimGrid  = dim3(blocksInX, blocksInY*blocksInZ);
    dim3 dimBlock = dim3(tBlock_x, tBlock_y, tBlock_z);

    // Note: the DVF and input image are passed via texture memory
    //-------------------------------------
    kernel <<< dimGrid, dimBlock >>> ( dev_output_vol,
                                       make_int3(output_vol_dim[0], output_vol_dim[1], output_vol_dim[2]),
                                       blocksInY );
    }
  else
    {
    dim3 dimGrid  = dim3(blocksInX, blocksInY, blocksInZ);
    dim3 dimBlock = dim3(tBlock_x, tBlock_y, tBlock_z);

    // Note: the DVF and input image are passed via texture memory
    //-------------------------------------
    kernel_3Dgrid <<< dimGrid, dimBlock >>> ( dev_output_vol,
                                              make_int3(output_vol_dim[0], output_vol_dim[1], output_vol_dim[2]));
    }

  CUDA_CHECK_ERROR;

  // Unbind the image and projection matrix textures
  hipUnbindTexture (tex_xdvf);
  hipUnbindTexture (tex_ydvf);
  hipUnbindTexture (tex_zdvf);
  hipUnbindTexture (tex_input_vol);
  CUDA_CHECK_ERROR;
  hipUnbindTexture (tex_IndexOutputToPPOutputMatrix);
  hipUnbindTexture (tex_IndexOutputToIndexDVFMatrix);
  hipUnbindTexture (tex_PPInputToIndexInputMatrix);
  CUDA_CHECK_ERROR;

  // Cleanup
  hipFreeArray ((hipArray*)array_xdvf);
  hipFreeArray ((hipArray*)array_ydvf);
  hipFreeArray ((hipArray*)array_zdvf);
  hipFreeArray ((hipArray*)array_input_vol);
  CUDA_CHECK_ERROR;
  hipFree (dev_IndexOutputToPPOutput);
  hipFree (dev_IndexOutputToIndexDVF);
  hipFree (dev_PPInputToIndexInput);
  CUDA_CHECK_ERROR;
}
